#include "hip/hip_runtime.h"
#define n 1024
#include<stdio.h>

__global__ void Matmul(int *d1,int *d2, int *d3)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int sum=0;
  for(int i=0; i<n; i++)
  {
   sum= sum + (d1[row*n+i]*d2[i*n+col]);
  }
  d3[row*n+col]=sum;
 }
int main()
{
int *a, *b, *c, i,j;
 a= new int[n*n];
 b= new int[n*n];
 c= new int[n*n];
 for(i=0;i<n;i++)
  {
   for(j=0;j<n;j++)
   {
    a[i*n+j]=1;
    b[i*n+j]=2;
   }
  }
 float t1= clock();
 int *dmat1, *dmat2, *dmat3;

  hipMalloc((void**)&dmat1,(n*n)*sizeof(int));
  hipMalloc((void**)&dmat2,(n*n)*sizeof(int));
  hipMalloc((void**)&dmat3,(n*n)*sizeof(int));

  
 dim3 threadsperblock(64,64);
 dim3 blockspergrid(16,16);
 hipMemcpy(dmat1, a, ((n*n)*sizeof(int)), hipMemcpyHostToDevice);
 hipMemcpy(dmat2, b, ((n*n)*sizeof(int)), hipMemcpyHostToDevice);
 
 Matmul<<<blockspergrid,threadsperblock>>>(dmat1,dmat2,dmat3);

 hipMemcpy(c, dmat3, ((n*n)*sizeof(int)), hipMemcpyDeviceToHost);
 float t2 = clock();
 float t3=(t2-t1);
 
 for(i=0;i<1;i++)
  {
   printf("\n");
   for(j=0;j<1;j++)
   {
    printf("%d\n",c[i*n+j]);
   }
  }
  printf("\n%f\n",(t3/1000000));
 hipFree(dmat1);
 hipFree(dmat2);
 hipFree(dmat3);
 free(a);free(b);free(c);
}
