#include "hip/hip_runtime.h"
#include<stdio.h>
#define n 512

__global__ void matinv( int *d1,int *d2)
{
 int row= blockIdx.y * blockDim.y + threadIdx.y;
 int col = blockIdx.x * blockDim.x + threadIdx.x;
   
 d2[col*n+row]=d1[row*n+col];
}

int main()
{
 int *a, *b;
 int *dmat1, *dmat2;
 a=(int*)malloc(n*n*sizeof(int));
 b=(int*)malloc(n*n*sizeof(int));
 float t1,t2,t3;
 int i,j;
 for(i=0;i<n;i++)
 {
  for(j=0;j<n;j++)
  {
   a[i*n+j]=j+1;
  }
 }
 
 t1=clock();
 hipMalloc((void**)&dmat1,n*n*sizeof(int));
 hipMalloc((void**)&dmat2,n*n*sizeof(int));
 
 hipMemcpy(dmat1, a,((n*n)*sizeof(int)),hipMemcpyHostToDevice);
 dim3 threadsperblock(2,2);
 dim3 blockspergrid(128,128);
 
 matinv<<<blockspergrid,threadsperblock>>>(dmat1,dmat2);

 hipMemcpy(b, dmat2, ((n*n)*sizeof(int)), hipMemcpyDeviceToHost);
 t2=clock();
 t3=(t2-t1);

 /*for(i=0;i<n;i++)
 {
  printf("\n");
  for(j=0;j<n;j++)
  {
   printf("\t%d",b[i*n+j]);
  }
 }*/
 printf("\n%f",(t3/1000000));
 hipFree(dmat1); hipFree(dmat2);free(a);free(b);
}
